#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 16  


__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N]) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    
    if (i < N && j < N) {
        C[i][j] = A[i][j] + B[i][j];
    }
}


int main() {
    float A[N][N], B[N][N], C[N][N];
    float (*d_A)[N], (*d_B)[N], (*d_C)[N];


    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i][j] = i * 0.1f + j * 0.1f;
            B[i][j] = j * 0.2f + i * 0.2f;
        }
    }


    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));


    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);


    dim3 threadsPerBlock(N, N);
    int numBlocks = 1;

    MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);


    printf("Result matrix C:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%0.1f ", C[i][j]);
        }
        printf("\n");
    }


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
